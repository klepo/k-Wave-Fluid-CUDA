/**
 * @file        CUDADeviceConstants.cu
 * @author      Jiri Jaros \n
 *              Faculty of Information Technology \n
 *              Brno University of Technology \n
 *              jarosjir@fit.vutbr.cz
 *
 * @brief       The implementation file for the class for storing constants residing in CUDA constant memory.
 *
 * @version     kspaceFirstOrder3D 3.4
 * @date        17 February 2016, 10:53 (created) \n
 *              12 April    2016, 15:02 (revised)
 *
 * @section License
 * This file is part of the C++ extension of the k-Wave Toolbox
 * (http://www.k-wave.org).\n Copyright (C) 2014 Jiri Jaros, Beau Johnston
 * and Bradley Treeby
 *
 * This file is part of the k-Wave. k-Wave is free software: you can
 * redistribute it and/or modify it under the terms of the GNU Lesser General
 * Public License as published by the Free Software Foundation, either version
 * 3 of the License, or (at your option) any later version.
 *
 * k-Wave is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for
 * more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with k-Wave. If not, see http://www.gnu.org/licenses/.
 */

#include <string>
#include <stdexcept>

#include <Parameters/CUDADeviceConstants.cuh>
#include <Utils/ErrorMessages.h>


//----------------------------------------------------------------------------//
//-------------------------------- Constants ---------------------------------//
//----------------------------------------------------------------------------//


//----------------------------------------------------------------------------//
//-------------------------------- Variables ---------------------------------//
//----------------------------------------------------------------------------//



/**
 * @variable CUDADeviceConstants
 * @brief    This variable holds basic simulation constants for GPU.
 * @details  This variable holds necessary simulation constants in the CUDA GPU.
 *           memory. This variable is imported as extern into other CUDA units
 */
__constant__ TCUDADeviceConstants CUDADeviceConstants;


//----------------------------------------------------------------------------//
//----------------------------- Global routines ------------------------------//
//----------------------------------------------------------------------------//


//----------------------------------------------------------------------------//
//---------------------------------- Public ----------------------------------//
//----------------------------------------------------------------------------//

__host__ void TCUDADeviceConstants::SetUpCUDADeviceConstatns()
{
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CUDADeviceConstants), this, sizeof(TCUDADeviceConstants)));
}// end of SetUpCUDADeviceConstatns
//------------------------------------------------------------------------------